// #include <cuda.h>
// #include <stdio.h>
// #include <stdlib.h>

// __global__ void mandelKernel() {
//     // To avoid error caused by the floating number, use the following pseudo code
//     //
//     // float x = lowerX + x * stepX;
//     // float y = lowerY + y * stepY;
// }

// // Host front-end function that allocates the memory and launches the GPU kernel
// void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
// {
//     float stepX = (upperX - lowerX) / resX;
//     float stepY = (upperY - lowerY) / resY;
// }

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 16

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int* img, int resX, int maxIterations) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + x * stepX;
    float c_im = lowerY + y * stepY;
    float z_re = c_re, z_im = c_im;

    int i;
    for (i = 0; i < maxIterations; ++i){

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    img[x + y * resX] = i;

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    size_t size = sizeof(int) * resX * resY;
    int* output;
    int* d_img;
    // output =  (int*)malloc(size);
    hipHostAlloc (&output, size, hipHostMallocDefault);
    hipMalloc(&d_img, size);

    dim3 threadsPerBlock(N, N);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);

    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, maxIterations);

    hipMemcpy(output, d_img, size, hipMemcpyDeviceToHost);
    memcpy(img, output, size);

    hipFree(d_img);
    hipHostFree(output);
}
