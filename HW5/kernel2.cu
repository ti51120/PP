#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BlockSize 8

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int* img, int resX, int maxIterations, int pitch) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + x * stepX;
    float c_im = lowerY + y * stepY;
    float z_re = c_re, z_im = c_im;

    int i;
    for (i = 0; i < maxIterations; ++i){

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    int* rowHead = (int*)((char*)img + y * pitch);
    rowHead[x] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    size_t size = sizeof(int) * resX * resY;
    size_t widthByte = sizeof(int) * resX;
    size_t height = resY;
    size_t pitch;
    int* output;
    int* d_img;
    
    hipHostAlloc (&output, size, hipHostMallocDefault);
    hipMallocPitch(&d_img, &pitch, widthByte, height);

    dim3 threadsPerBlock(BlockSize, BlockSize);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, maxIterations, pitch);

    hipMemcpy2D(output, widthByte, d_img, pitch, widthByte, height, hipMemcpyDeviceToHost);
    memcpy(img, output, size);

    hipFree(d_img);
    hipHostFree(output);
}